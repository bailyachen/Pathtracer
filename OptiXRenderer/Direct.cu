#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

#define PI 3.14159265359

#define MIDPOINT 0.5 

rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(int1, reduceVariance, , );
rtDeclareVariable(int1, lightSamples, ,);
rtDeclareVariable(int1, lightStratify, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

float3 f(float3 w_i, float3 w_o, float3 n, float3 k_d, float3 k_s, float s) {
    float3 r = reflect(-w_o, n);
    return (k_d / PI) + k_s * ((s + 2) / (2 * PI)) * pow(fmaxf(dot(r, w_i), 0), s);
}

float G(float3 x, float3 x_prime, float3 surface_n, float3 light_n) {
    float R = length(x_prime - x);
    float3 w_i = normalize(x_prime - x);
    return (1 / (R * R)) * fmaxf(dot(surface_n, w_i), 0) * fmaxf(dot(light_n, w_i), 0);
}

RT_PROGRAM void closestHit() {

    MaterialValue mv = attrib.mv;

    float3 result = mv.ambient + mv.emission;

    Config cf = config[0];
    int N = lightSamples.x;

    int stratify = lightStratify.x;

    float3 x = attrib.intersection;
    float3 n = attrib.normal;
    float3 w_o = attrib.wo;

    for(int i = 0; i < qlights.size(); i++) {
        QuadLight qlight = qlights[i];

        float3 l_d = make_float3(0);
        
        float3 n_l = normalize(cross(qlight.ab, qlight.ac));
        float A = length(cross(qlight.ab, qlight.ac));

        if(stratify) {
            float sqrtN = sqrtf((float) N);

            float3 dAB = qlight.ab / sqrtN;
            float3 dAC = qlight.ac / sqrtN;

            for(int j = 0; j < sqrtN; j++) {
                for(int k = 0; k < sqrtN; k++) {
                    float u1 = reduceVariance.x ? MIDPOINT : rnd(payload.seed);
                    float u2 = reduceVariance.x ? MIDPOINT : rnd(payload.seed);

                    float3 x_prime = qlight.a + (j * dAB) + (k * dAC) + (u1 * dAB) + (u2 * dAC);
                    float3 w_i = normalize(x_prime - x);

                    float R = length(x_prime - x);

                    // Visibility
                    ShadowPayload shadowPayload;
                    shadowPayload.isVisible = true;

                    Ray shadowRay = make_Ray(x + (cf.epsilon * w_i), w_i, 1, cf.epsilon, R - 2 * cf.epsilon);
                    
                    rtTrace(root, shadowRay, shadowPayload);

                    if(shadowPayload.isVisible) {
                        l_d += f(w_i, w_o, n, mv.diffuse, mv.specular, mv.shininess) * G(x, x_prime, n, n_l);
                    }
                }
            }
        } else {
            for(int k = 0; k < N; k++) {
                float u1 = rnd(payload.seed);
                float u2 = rnd(payload.seed);

                float3 x_prime = qlight.a + (u1 * qlight.ab) + (u2 * qlight.ac);
                float3 w_i = normalize(x_prime - x);

                float R = length(x_prime - x);

                // Visibility
                ShadowPayload shadowPayload;
                shadowPayload.isVisible = true;

                Ray shadowRay = make_Ray(x + (cf.epsilon * w_i), w_i, 1, cf.epsilon, R - 2 * cf.epsilon);
                
                rtTrace(root, shadowRay, shadowPayload);

                if(shadowPayload.isVisible) {
                    l_d += f(w_i, w_o, n, mv.diffuse, mv.specular, mv.shininess) * G(x, x_prime, n, n_l);
                }
            }
        }

        l_d *= qlight.intensity * A / N;

        result += l_d;
    }

    payload.radiance = result;
}