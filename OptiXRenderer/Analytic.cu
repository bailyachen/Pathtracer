#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

#define PI 3.14159265359

rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit() {

    float3 result = attrib.mv.ambient + attrib.mv.emission;
    float3 r = attrib.intersection;

    // Diffuse Albedo
    float3 f = attrib.mv.diffuse / PI;

    float3 E = make_float3(0, 0, 0);

    for(int i = 0; i < qlights.size(); i++) {

        float3 v[] = { 
            qlights[i].a,                                 // A
            qlights[i].a + qlights[i].ab,                 // B
            qlights[i].a + qlights[i].ab + qlights[i].ac, // D
            qlights[i].a + qlights[i].ac                  // C
        };
        

        float3 irradiance = make_float3(0, 0, 0);
        

        for(int k = 0; k < 4; k++) {
            int next = (k + 1) % 4;
            float Theta_k = acos(dot(normalize(v[k] - r), normalize(v[next] - r)));
            float3 Gamma_k = normalize(cross(v[k] - r, v[next] - r));

            irradiance += Gamma_k * Theta_k;
        }

        irradiance *= 0.5;

        E += qlights[i].intensity * dot(irradiance, attrib.normal);
    }


    result +=  f * E;
    payload.radiance = result;
}