#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

#define PI 3.14159265359

// Sampling Modes
#define HEMISPHERE 0
#define COSINE 1
#define BRDF 2

#define PHONG 0
#define GGX 1

#define OFF 0
#define ON 1
#define MIS 2

rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtDeclareVariable(int1, NEE, , );
rtDeclareVariable(int1, RR, ,);
rtDeclareVariable(int1, importanceSampling, , );

rtBuffer<Config> config; // Config

// Declare attributes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

/*************************************************************
        RANDOM SAMPLING FUNCTIONS
*************************************************************/

float3 random_hemisphere_vector(unsigned int seed, float3 n) {
    float r0 = rnd(seed), r1 = rnd(seed);

    float theta = acos(r0);
    float phi = 2 * PI * r1;

    float3 s = make_float3(cos(phi) * sin(theta), sin(phi) * sin(theta), cos(theta));

    float3 w = normalize(n);
    float3 a = make_float3(1, 1, 1);

    float3 u = normalize(cross(a, w));
    float3 v = cross(w, u);

    return (s.x * u) + (s.y * v) + (s.z * w); 
}

float3 random_cosine_vector(unsigned int seed, float3 n) {
    float r0 = rnd(seed), r1 = rnd(seed);

    float theta  = acos(sqrt(r0));
    float phi = 2 * PI * r1;
    
    float3 s = make_float3(cos(phi) * sin(theta), sin(phi) * sin(theta), cos(theta));

    float3 w = normalize(n);
    float3 a = make_float3(1, 1, 1);

    float3 u = normalize(cross(a, w));
    float3 v = cross(w, u);

    return (s.x * u) + (s.y * v) + (s.z * w); 
}

float3 random_phong_vector(unsigned int seed, float3 w_o, float3 n, MaterialValue mv) {
    float3 k_s = mv.specular;
    float _k_s = (k_s.x + k_s.y + k_s.z) / 3.0f;
    float3 k_d = mv.diffuse;
    float _k_d = (k_d.x + k_d.y + k_d.z) / 3.0f;
    float t = _k_s / (_k_d + _k_s);

    float r0 = rnd(seed), r1 = rnd(seed), r2 = rnd(seed);

    float phi = 2 * PI * r2;
    float theta;

    float3 u, v, w; // Coordinate Basis

    if(r0 <= t) { // Specular
        float3 r = reflect(-w_o, n);
        w = normalize(r);
        theta = acos(pow(r1, 1 / (mv.shininess + 1)));
    }
    else { // Diffuse
        w = normalize(n);
        theta = acos(sqrt(r1));
    }

    float3 s = make_float3(cos(phi) * sin(theta), sin(phi) * sin(theta), cos(theta));

    float3 a = make_float3(1, 1, 1);
    
    u = normalize(cross(a, w));
    v = cross(w, u);

    return (s.x * u) + (s.y * v) + (s.z * w);
}

float3 random_GGX_vector(unsigned int seed, float3 w_o, float3 n, MaterialValue mv) {
    float3 k_s = mv.specular;
    float _k_s = (k_s.x + k_s.y + k_s.z) / 3.0f;
    float3 k_d = mv.diffuse;
    float _k_d = (k_d.x + k_d.y + k_d.z) / 3.0f;
    float t = _k_d == 0 && _k_s == 0 ? 1 : fmaxf(0.25, _k_s / (_k_d + _k_s));

    float r0 = rnd(seed), r1 = rnd(seed), r2 = rnd(seed);

    float3 s;

    float phi = 2 * PI * r2;
    float theta;

    float3 u, v, w; // Coordinate Basis

    if(r0 <= t) { // Specular
        w = normalize(n);
        theta = atan2(mv.roughness * sqrt(r1), sqrt(1 - r1));
        float3 h = make_float3(cos(phi) * sin(theta), sin(phi) * sin(theta), cos(theta)); // Half Vector

        float3 a = make_float3(1, 1, 1);
    
        u = normalize(cross(a, w));
        v = normalize(cross(w, u));

        h = normalize((h.x * u) + (h.y * v) + (h.z * w));

        s = reflect(-w_o, h);
    } else { // Diffuse
        w = normalize(n);
        theta = acos(sqrtf(r1));
        s = make_float3(cos(phi) * sin(theta), sin(phi) * sin(theta), cos(theta));

        float3 a = make_float3(1, 1, 1);
    
        u = normalize(cross(a, w));
        v = cross(w, u);
        s = (s.x * u) + (s.y * v) + (s.z * w);
    }

    return s;
}
/*************************************************************
        PHONG BRDF FUNCTIONS
*************************************************************/

float3 f_phong(float3 w_i, float3 w_o, float3 n, MaterialValue mv) {
    float3 r = reflect(-w_o, n);

    float3 k_d = mv.diffuse;
    float3 k_s = mv.specular;
    float s = mv.shininess;

    return (k_d / PI) + k_s * ((s + 2) / (2 * PI)) * pow(fmaxf(dot(r, w_i), 0), s);
}

float G_phong(float3 x, float3 x_prime, float3 surface_n, float3 light_n) {
    float R = length(x_prime - x);
    float3 w_i = normalize(x_prime - x);
    return (1 / (R * R)) * fmaxf(dot(surface_n, w_i), 0) * fmaxf(dot(light_n, w_i), 0);
}


float PDF_phong(float3 w_i, float3 w_o, float3 n, MaterialValue mv) {

    float3 k_s = mv.specular;
    float _k_s = (k_s.x + k_s.y + k_s.z) / 3.0f;
    float3 k_d = mv.diffuse;
    float _k_d = (k_d.x + k_d.y + k_d.z) / 3.0f;
    float s = mv.shininess;

    float t =_k_d == 0 && _k_s == 0 ? 1 : _k_s / (_k_d + _k_s);

    float3 r = reflect(-w_o, n);

    float result =  ((1 - t) * fmaxf(dot(n, w_i), 0) / PI);
    result += (t * (s + 1) / (2 * PI) * pow(fmaxf(dot(r, w_i), 0), s));

    return result;
}

/*************************************************************
        GGX BRDF FUNCTIONS
*************************************************************/

float power(float x, int n) {
    int zerodir;
    float factor;

    if(n < 0) {
        zerodir = 1;
        factor = 1.0f / x;
    } else {
        zerodir = -1;
        factor = x;
    }

    float result = 1;
    while(n) {
        if(n & 1) { // n % 2 != 0
            result *= factor;
            n += zerodir;
        } else {
            factor *= factor;
            n >>= 1; // n /= 2;
        }
    }
    return result;
}

float3 Fresnel(float3 w_i, float3 h, float3 n, MaterialValue mv) {
    float3 k_s = mv.specular;
    return k_s + (1 - k_s) * power(1 - clamp(dot(h, w_i), 0.0f, 1.0f), 5);
}

float G_1(float3 v, float3 n, MaterialValue mv) {
    if (dot(v, n) > 0) {
        float theta_v = acos(clamp(dot(v, n), 0.0f, 1.0f));
        float alpha = mv.roughness;
        float tan_theta_v = tan(theta_v);

        return 2 / (1 + sqrt(1 + power(alpha, 2) * power(tan_theta_v, 2)));
    }
    return 0;
}

float G_GGX(float3 w_i, float3 w_o, float3 n, MaterialValue mv) {
    return G_1(w_i, n, mv) * G_1(w_o, n, mv);
}

float D(float3 h, float3 n, MaterialValue mv) {
    float alpha_2 = power(mv.roughness, 2);

    float theta_h = acos(clamp(dot(h, n), -1.0f, 1.0f));
    
    float tan_theta_h_2 = power(tan(theta_h), 2);
    float cos_theta_h_4 = power(cos(theta_h), 4);

    float alphatan_2 = power(alpha_2 + tan_theta_h_2, 2);

    float result =  alpha_2 / (PI * cos_theta_h_4 * alphatan_2);

    return result;
}

float3 _f_ggx(float3 w_i, float3 w_o, float3 n, MaterialValue mv) {
    float3 h = normalize(w_i + w_o);

    float n_dot_wi = clamp(dot(n, w_i), 0.0f, 1.0f);
    float n_dot_wo = clamp(dot(n, w_o), 0.0f, 1.0f);

    if(n_dot_wi <= 0 || n_dot_wo <= 0)
        return make_float3(0);

    float3 result = make_float3(1);
    result *= Fresnel(w_i, h, n, mv);
    result *= G_GGX(w_i, w_o, n, mv);
    result *= D(h, n, mv);
    result /= 4 * n_dot_wi * n_dot_wo;

    return result;
}

float3 f_GGX(float3 w_i, float3 w_o, float3 n, MaterialValue mv) {
    float3 k_d = mv.diffuse;
    return k_d / PI + _f_ggx(w_i, w_o, n, mv);
}

float PDF_GGX(float3 w_i, float3 w_o, float3 n, MaterialValue mv) {
    float3 k_s = mv.specular;
    float _k_s = (k_s.x + k_s.y + k_s.z) / 3.0f;
    float3 k_d = mv.diffuse;
    float _k_d = (k_d.x + k_d.y + k_d.z) / 3.0f;

    float t = _k_d + _k_s <= 0 ? 1 : fmaxf(0.25, _k_s / (_k_d + _k_s));
    float3 h = normalize(w_i + w_o);
    float d = D(h, n, mv);
    
    float result = ((1 - t) * clamp(dot(n, w_i), 0.0f, 1.0f) / PI);
    result += t * d * clamp(dot(n, h), 0.0f, 1.0f) / (4 * clamp(dot(h, w_i), 0.0f, 1.0f));

    return result;
}

/*************************************************************
        LIGHTING
*************************************************************/

float3 directLight(float3 x, float3 n, float3 w_o, MaterialValue mv, Config cf) {
    float3 result = make_float3(0);
    for (int i = 0; i < qlights.size(); i++) {
        QuadLight qlight = qlights[i];

        float3 l_d = make_float3(0);

        float3 n_l = normalize(cross(qlight.ab, qlight.ac));
        float A = length(cross(qlight.ab, qlight.ac));

        float u1 = rnd(payload.seed);
        float u2 = rnd(payload.seed);

        float3 x_prime = qlight.a + (u1 * qlight.ab) + (u2 * qlight.ac);
        float3 w_i = normalize(x_prime - x);

        float R = length(x_prime - x);

        // Visibility
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;

        Ray shadowRay = make_Ray(x + (cf.epsilon * w_i), w_i, 1, cf.epsilon, R - 2 * cf.epsilon);

        rtTrace(root, shadowRay, shadowPayload);

        if (shadowPayload.isVisible) {
            float3 f;
            if(mv.BRDFmode == GGX) {
                 f = f_GGX(w_i, w_o, n, mv);
            } else {
                f = f_phong(w_i, w_o, n, mv);
            }
            
            l_d += f * G_phong(x, x_prime, n, n_l);
        }

        l_d *= qlight.intensity * A;

        result += l_d;
    }
    return result;
}

float PDF_BRDF(float3 w_i, float3 w_o, float3 n, MaterialValue mv) {
    switch(importanceSampling.x) {
        case HEMISPHERE:
            return 1 / (2 * PI);
        case BRDF:
            switch(mv.BRDFmode) {
                case GGX:
                    return PDF_GGX(w_i, w_o, n, mv);
                case PHONG:
                default:
                    return PDF_phong(w_i, w_o, n, mv);
            }
        case COSINE:
        default:
            return clamp(dot(n, w_i), 0.0f, 1.0f) / PI;
    }
}

float PDF_NEE(unsigned int seed, float3 x, float3 w_i, Config cf) {
    float pdf = 0;

    for(int i = 0; i < qlights.size(); i++) {
        QuadLight qlight = qlights[i];

        Payload nPayload;
        nPayload.radiance = make_float3(0);
        nPayload.throughput = make_float3(1);
        nPayload.done = true;
        nPayload.depth = 1;
        nPayload.seed = seed;

        Ray ray = make_Ray(x + cf.epsilon * w_i, w_i, 0, cf.epsilon, RT_DEFAULT_MAX);
        
        rtTrace(root, ray, nPayload);

        float3 l_e = nPayload.radiance;
        float pdf_l = 0;

        if(length(l_e) > 0) {
            float3 x_prime = nPayload.intersection;
            float R = length(x_prime - x);
            float A = length(cross(qlight.ab, qlight.ac));
            float3 n_l = normalize(cross(qlight.ab, qlight.ac));

            if(dot(n_l, w_i) > 0) {
                pdf_l = power(R, 2) / (A * dot(n_l, w_i));
            }
        }

        pdf += pdf_l;
    }

    return pdf / qlights.size();
}

float3 direct_BRDF(unsigned int seed, float3 x, float3 w_o, float3 n, MaterialValue mv, Config cf) {
    float3 w_i;
    switch(importanceSampling.x) {
        case HEMISPHERE:
            w_i = random_hemisphere_vector(seed, n);
            break;
        case BRDF:
            switch(mv.BRDFmode) {
                case GGX:
                    w_i = random_GGX_vector(seed, w_o, n, mv);
                    break;
                case PHONG:
                default:
                    w_i = random_phong_vector(seed, w_o, n, mv);
                    break;
            }
            break;
        case COSINE:
        default:
            w_i = random_cosine_vector(seed, n);
            break;
    }

    float3 f;
    switch(mv.BRDFmode) {
        case GGX:
            f = f_GGX(w_i, w_o, n, mv);
            break;
        case PHONG:
        default:
            f = f_phong(w_i, w_o, n, mv);
            break;
    }

    float pdf = PDF_BRDF(w_i, w_o, n, mv);

    if(pdf == 0) return make_float3(0);

    float pdf_BRDF = PDF_BRDF(w_i, w_o, n, mv);
    float pdf_NEE = PDF_NEE(seed, x, w_i, cf);
    float weight = (pdf_BRDF * pdf_BRDF) / (pdf_BRDF * pdf_BRDF + pdf_NEE * pdf_NEE);

    Payload nPayload;
    nPayload.radiance = make_float3(0);

    nPayload.throughput = weight * f * clamp(dot(n, w_i), 0.0f, 1.0f) / pdf;
    nPayload.done = true;
    nPayload.depth = 1;
    nPayload.seed = seed;

    Ray ray = make_Ray(x + cf.epsilon * w_i, w_i, 0, cf.epsilon, RT_DEFAULT_MAX);
    rtTrace(root, ray, nPayload);

    return nPayload.radiance;
}

float3 direct_NEE(unsigned int seed, float3 x, float3 w_o, float3 n, MaterialValue mv, Config cf) {
    float3 result = make_float3(0);

    for(int i = 0; i < qlights.size(); i++) {
        QuadLight qlight = qlights[i];

        float u1 = rnd(seed), u2 = rnd(seed);
        float3 x_prime = qlight.a + (u1 * qlight.ab) + (u2 * qlight.ac);
        float3 w_i = normalize(x_prime - x);
        float R = length(x_prime - x);

        float3 f = make_float3(0);

        float pdf_l = 0;

        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;

        Ray shadowRay = make_Ray(x + cf.epsilon * w_i, w_i, 1, cf.epsilon, R - (2 * cf.epsilon));
        rtTrace(root, shadowRay, shadowPayload);

        float3 l_d = make_float3(0);

        if(shadowPayload.isVisible) {
            switch(mv.BRDFmode) {
                case GGX:
                    f = f_GGX(w_i, w_o, n, mv);
                    break;
                case PHONG:
                default:
                    f = f_phong(w_i, w_o, n, mv);
                    break;
            }

            float A = length(cross(qlight.ab, qlight.ac));
            float3 n_l = normalize(cross(qlight.ab, qlight.ac));

            if(dot(n_l, w_i) > 0) {
                pdf_l = power(R, 2) / (A * dot(n_l, w_i));
            } else {
                pdf_l = 0;
            }
        }

        float pdf_BRDF = PDF_BRDF(w_i, w_o, n, mv);
        float pdf_NEE = PDF_NEE(seed, x, w_i, cf);
        float weight = pdf_NEE * pdf_NEE / (pdf_BRDF * pdf_BRDF + pdf_NEE * pdf_NEE);

        if(pdf_l != 0) {
            l_d = weight * qlight.intensity * f * clamp(dot(n, w_i), 0.0f, 1.0f) / pdf_l;
            result += l_d;
        }
    }

    return result;
}

RT_PROGRAM void closestHit() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = make_float3(0);

    float3 x = attrib.intersection;
    float3 n = attrib.normal;
    float3 w_o = attrib.wo;

    bool isEmissive = length(mv.emission) > 0;
    // NEE
    if(NEE.x == ON) {
        if(isEmissive) {
            payload.done = true;

            if (payload.depth == 0)
                payload.radiance = mv.emission;

            return;
        }

        float3 direct = directLight(x, n, w_o, mv, cf);

        payload.radiance += direct * payload.throughput;
    } else if (NEE.x == MIS) {
        if(payload.done) {
            if(dot(w_o, n) > 0) {
                payload.radiance = mv.emission * payload.throughput;

            } else {
                payload.radiance = make_float3(0);
            }
            payload.intersection = x;
            return;
        }
        if(payload.depth == 0) {
            if(dot(n, w_o) < 0) {
                result += mv.emission;
            }
        }
        if(isEmissive) {
            payload.radiance = result * payload.throughput;
            payload.done = true;
            return;
        }

        result += direct_BRDF(payload.seed, x, w_o, n, mv, cf);
        result += direct_NEE(payload.seed, x, w_o, n, mv, cf);

        payload.radiance = result * payload.throughput;
    } else {
        if(fmaxf(dot(n, w_o), 0) > 0)
            result += mv.emission;

        if(isEmissive) {
            payload.radiance = result * payload.throughput;
            payload.done = true;
            return;
        }
    }

    // RR
    if(RR.x) {
        float q = 1.f - fminf(fmaxf(payload.throughput), 1.f); // Probability of Termination
        float r = rnd(payload.seed);

        if(r <= q) {
            payload.done = true;
        } else {
            payload.throughput *= 1.f / (1.f - q);
        }
    }

    float3 w_i = make_float3(0, 1, 0); // Always UP
    float pdf = 0;

    switch(importanceSampling.x) {
        case HEMISPHERE:
            w_i = random_hemisphere_vector(payload.seed, n);
            payload.throughput *= 2 * PI * f_phong(w_i, w_o, n, mv) * fmaxf(dot(n, w_i), 0);
            break;
        case BRDF:
            switch(mv.BRDFmode) {
                case GGX:
                    w_i = random_GGX_vector(payload.seed, w_o, n, mv);
                    pdf = PDF_GGX(w_i, w_o, n, mv);
                    if(pdf == 0) {
                        payload.radiance = make_float3(0);
                        payload.done = true;
                    } else {
                        payload.throughput *= f_GGX(w_i, w_o, n, mv) * fmaxf(dot(n, w_i), 0) / pdf;
                    }
                    break;
                case PHONG:
                default:
                    w_i = random_phong_vector(payload.seed, w_o, n, mv);
                    payload.throughput *= f_phong(w_i, w_o, n, mv) * fmaxf(dot(n, w_i), 0) / PDF_phong(w_i, w_o, n, mv);
                    break;
            }
            break;
        case COSINE:
        default:
            w_i = random_cosine_vector(payload.seed, n);
            payload.throughput *= PI * f_phong(w_i, w_o, n, mv);
            break;
    }

    payload.origin = x;
    payload.dir = w_i; 
    payload.depth++;
}
